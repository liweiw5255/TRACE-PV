#include "hip/hip_runtime.h"
// multi_gpu.cu
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iostream>

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        std::cerr << "CUDA error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void kernel(int gpu_id) {
    // Simple kernel to run on each GPU
    printf("Hello from GPU %d\n", gpu_id);
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);
    
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Get the number of GPUs
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    checkCUDAError("hipGetDeviceCount");

    if (rank < deviceCount) {
        // Assign GPU to each process
        hipSetDevice(rank);
        checkCUDAError("hipSetDevice");

        // Launch kernel
        kernel<<<1, 1>>>(rank);
        checkCUDAError("kernel launch");

        hipDeviceSynchronize();
        checkCUDAError("hipDeviceSynchronize");
    } else {
        std::cerr << "Rank " << rank << " exceeds device count " << deviceCount << std::endl;
    }

    MPI_Finalize();
    return 0;
}